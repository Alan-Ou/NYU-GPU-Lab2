/*
 * This file contains the code for doing the heat distribution problem.
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s), memory allocation, data movement, etc.
 *
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

/* To index element (i,j) of a 2D square array of dimension NxN stored as 1D
   index(i, j, N) means access element at row i, column j, and N is the dimension which is NxN */
#define index(i, j, N) ((i) * (N)) + (j)

/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void seq_heat_dist(float *, unsigned int, unsigned int);
void gpu_heat_dist(float *, unsigned int, unsigned int);
__global__ void kernel_logic(float *, unsigned int);
void print_matrix(float *, unsigned int);

/*****************************************************************/
/**** Do NOT CHANGE ANYTHING in main() function ******/

int main(int argc, char *argv[])
{
  unsigned int N;         /* Dimention of NxN matrix */
  int type_of_device = 0; // CPU or GPU
  int iterations = 0;
  int i;

  /* The 2D array of points will be treated as 1D array of NxN elements */
  float *playground;

  // to measure time taken by a specific part of the code
  double time_taken;
  clock_t start, end;

  if (argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU version\n");
    exit(1);
  }

  type_of_device = atoi(argv[3]);
  N = (unsigned int)atoi(argv[1]);
  iterations = (unsigned int)atoi(argv[2]);

  /* Dynamically allocate NxN array of floats */
  playground = (float *)calloc(N * N, sizeof(float));
  if (!playground)
  {
    fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
    exit(1);
  }

  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements  initialization
  for (i = 0; i < N; i++)
    playground[index(0, i, N)] = 100;
  for (i = 0; i < N; i++)
    playground[index(N - 1, i, N)] = 150;
  for (i = 1; i < N - 1; i++)
    playground[index(i, 0, N)] = 80;
  for (i = 1; i < N - 1; i++)
    playground[index(i, N - 1, N)] = 80;

  switch (type_of_device)
  {
  case 0:
    printf("CPU sequential version:\n");
    start = clock();
    seq_heat_dist(playground, N, iterations);
    end = clock();
    break;

  case 1:
    printf("GPU version:\n");
    start = clock();
    gpu_heat_dist(playground, N, iterations);
    hipDeviceSynchronize();
    end = clock();
    break;

  default:
    printf("Invalid device type\n");
    exit(1);
  }

  time_taken = ((double)(end - start)) / CLOCKS_PER_SEC;

  printf("Time taken = %lf\n", time_taken);

  free(playground);

  return 0;
}

/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void seq_heat_dist(float *playground, unsigned int N, unsigned int iterations)
{
  // Loop indices
  int i, j, k;
  int upper = N - 1;

  // number of bytes to be copied between array temp and array playground
  unsigned int num_bytes = 0;

  float *temp;
  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  temp = (float *)calloc(N * N, sizeof(float));
  if (!temp)
  {
    fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
    exit(1);
  }

  num_bytes = N * N * sizeof(float);

  /* Copy initial array in temp */
  memcpy((void *)temp, (void *)playground, num_bytes);

  for (k = 0; k < iterations; k++)
  {
    /* Calculate new values and store them in temp */
    for (i = 1; i < upper; i++)
      for (j = 1; j < upper; j++)
        temp[index(i, j, N)] = (playground[index(i - 1, j, N)] +
                                playground[index(i + 1, j, N)] +
                                playground[index(i, j - 1, N)] +
                                playground[index(i, j + 1, N)]) /
                               4.0;

    /* Move new values into old values */
    memcpy((void *)playground, (void *)temp, num_bytes);
  }
  // output the computation results
  print_matrix(playground, N);
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kernels if you want ********************/
void gpu_heat_dist(float *playground, unsigned int N, unsigned int iterations)
{

  /* Here you have to write any cuda dynamic allocations, any communications between device and host, any number of kernel
     calls, etc. */

  float *temp;
  // memory allocation
  hipMalloc((void **)&temp, N * N * sizeof(float));

  // communication: host to device
  hipMemcpy(temp, playground, N * N * sizeof(float), hipMemcpyHostToDevice);

  // configuration
  dim3 sizeOfBlock(16, 16);
  dim3 sizeOfGrid((N + sizeOfBlock.x - 1) / sizeOfBlock.x, (N + sizeOfBlock.y - 1) / sizeOfBlock.y);

  for (int i = 0; i < iterations; i++)
  {
    kernel_logic<<<sizeOfGrid, sizeOfBlock>>>(temp, N);
    hipDeviceSynchronize();
  }

  // communication: device to host
  hipMemcpy(playground, temp, N * N * sizeof(float), hipMemcpyDeviceToHost);

  // output the computation results
  print_matrix(playground, N);

  // free the allocated memory
  hipFree(temp);
}

__global__ void kernel_logic(float *temp, unsigned int N)
{
  int row = blockIdx.x * blockDim.x + threadIdx.x;
  int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < 1 || row >= N - 1 || col < 1 || col >= N - 1)
  {
    return;
  }

  float update_value = (temp[index(row - 1, col, N)] +
                        temp[index(row + 1, col, N)] +
                        temp[index(row, col - 1, N)] +
                        temp[index(row, col + 1, N)]) /
                       4.0f;

  temp[index(row, col, N)] = update_value;
}

void print_matrix(float *matrix, unsigned int N)
{
  for (int row = 0; row < N; row++)
  {
    for (int col = 0; col < N; col++)
    {
      printf("%f ", matrix[index(row, col, N)]);
    }
    printf("\n");
  }
}
